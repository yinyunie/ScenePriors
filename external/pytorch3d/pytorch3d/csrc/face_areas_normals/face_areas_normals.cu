#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <tuple>

template <typename scalar_t>
__global__ void FaceAreasNormalsForwardKernel(
    const scalar_t* __restrict__ verts,
    const int64_t* __restrict__ faces,
    scalar_t* __restrict__ face_areas,
    scalar_t* __restrict__ face_normals,
    const size_t V,
    const size_t F) {
  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = gridDim.x * blockDim.x;

  // Faces split evenly over the number of threads in the grid.
  // Each thread computes the area & normal of its respective faces and adds it
  // to the global face_areas tensor.
  for (size_t f = tid; f < F; f += stride) {
    const int64_t i0 = faces[3 * f + 0];
    const int64_t i1 = faces[3 * f + 1];
    const int64_t i2 = faces[3 * f + 2];

    const scalar_t v0_x = verts[3 * i0 + 0];
    const scalar_t v0_y = verts[3 * i0 + 1];
    const scalar_t v0_z = verts[3 * i0 + 2];

    const scalar_t v1_x = verts[3 * i1 + 0];
    const scalar_t v1_y = verts[3 * i1 + 1];
    const scalar_t v1_z = verts[3 * i1 + 2];

    const scalar_t v2_x = verts[3 * i2 + 0];
    const scalar_t v2_y = verts[3 * i2 + 1];
    const scalar_t v2_z = verts[3 * i2 + 2];

    const scalar_t ax = v1_x - v0_x;
    const scalar_t ay = v1_y - v0_y;
    const scalar_t az = v1_z - v0_z;

    const scalar_t bx = v2_x - v0_x;
    const scalar_t by = v2_y - v0_y;
    const scalar_t bz = v2_z - v0_z;

    const scalar_t cx = ay * bz - az * by;
    const scalar_t cy = az * bx - ax * bz;
    const scalar_t cz = ax * by - ay * bx;

    scalar_t norm = sqrt(cx * cx + cy * cy + cz * cz);
    face_areas[f] = norm / 2.0;
    norm = (norm < 1e-6) ? 1e-6 : norm; // max(norm, 1e-6)
    face_normals[3 * f + 0] = cx / norm;
    face_normals[3 * f + 1] = cy / norm;
    face_normals[3 * f + 2] = cz / norm;
  }
}

// TODO(gkioxari) support all data types once AtomicAdd supports doubles.
// Currently, support is for floats only.
__global__ void FaceAreasNormalsBackwardKernel(
    const float* __restrict__ grad_areas,
    const float* __restrict__ grad_normals,
    const float* __restrict__ verts,
    const int64_t* __restrict__ faces,
    float* __restrict__ grad_verts,
    const size_t V,
    const size_t F) {
  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = gridDim.x * blockDim.x;

  // Faces split evenly over the number of threads in the grid.
  // Each thread computes the area & normal of its respective faces and adds it
  // to the global face_areas tensor.
  for (size_t f = tid; f < F; f += stride) {
    const int64_t i0 = faces[3 * f + 0];
    const int64_t i1 = faces[3 * f + 1];
    const int64_t i2 = faces[3 * f + 2];

    const float v0_x = verts[3 * i0 + 0];
    const float v0_y = verts[3 * i0 + 1];
    const float v0_z = verts[3 * i0 + 2];

    const float v1_x = verts[3 * i1 + 0];
    const float v1_y = verts[3 * i1 + 1];
    const float v1_z = verts[3 * i1 + 2];

    const float v2_x = verts[3 * i2 + 0];
    const float v2_y = verts[3 * i2 + 1];
    const float v2_z = verts[3 * i2 + 2];

    const float ax = v1_x - v0_x;
    const float ay = v1_y - v0_y;
    const float az = v1_z - v0_z;

    const float bx = v2_x - v0_x;
    const float by = v2_y - v0_y;
    const float bz = v2_z - v0_z;

    const float cx = ay * bz - az * by;
    const float cy = az * bx - ax * bz;
    const float cz = ax * by - ay * bx;

    float norm = sqrt(cx * cx + cy * cy + cz * cz);
    norm = (norm < 1e-6) ? 1e-6 : norm; // max(norm, 1e-6)
    float inv_norm = 1. / norm;
    float inv_norm_2 = pow(inv_norm, 2.0f);
    float inv_norm_3 = pow(inv_norm, 3.0f);

    // We compute gradients with respect to the input vertices.
    // For each vertex, gradients come from grad_areas and grad_normals.
    // eg, grad_v0_x = (d / d v0_x)
    //       = \sum_f (d / d areas[f]) * (d areas[f] / d v0_x)
    //              + (d / d normals[f, 0]) * (d normals[f, 0] / d v0_x)
    //              + (d / d normals[f, 1]) * (d normals[f, 1] / d v0_x)
    //              + (d / d normals[f, 2]) * (d normals[f, 2] / d v0_x)
    // with (d / d areas[f]) = grad_areas[f] and
    //      (d / d normals[f, j]) = grad_normals[f][j].
    // The equations below are derived after taking
    // derivatives wrt to the vertices (fun times!).

    // grad v0 coming from grad areas and grad normals
    const float grad_v0_x =
        ((-az + bz) * cy + (-by + ay) * cz) / 2.0 * inv_norm * grad_areas[f] +
        -cx * ((-az + bz) * cy + (-by + ay) * cz) * inv_norm_3 *
            grad_normals[3 * f + 0] +
        ((-az + bz) - cy * ((-az + bz) * cy + (-by + ay) * cz) * inv_norm_2) *
            inv_norm * grad_normals[3 * f + 1] +
        ((-by + ay) - cz * ((-az + bz) * cy + (-by + ay) * cz) * inv_norm_2) *
            inv_norm * grad_normals[3 * f + 2];
    atomicAdd(grad_verts + 3 * i0 + 0, grad_v0_x);

    const float grad_v0_y =
        ((-bz + az) * cx + (-ax + bx) * cz) / 2.0 * inv_norm * grad_areas[f] +
        ((-bz + az) - cx * ((-bz + az) * cx + (-ax + bx) * cz) * inv_norm_2) *
            inv_norm * grad_normals[3 * f + 0] +
        -cy * ((-bz + az) * cx + (-ax + bx) * cz) * inv_norm_3 *
            grad_normals[3 * f + 1] +
        ((-ax + bx) - cz * ((-bz + az) * cx + (-ax + bx) * cz) * inv_norm_2) *
            inv_norm * grad_normals[3 * f + 2];
    atomicAdd(grad_verts + 3 * i0 + 1, grad_v0_y);

    const float grad_v0_z =
        ((-ay + by) * cx + (-bx + ax) * cy) / 2.0 * inv_norm * grad_areas[f] +
        ((-ay + by) - cx * ((-ay + by) * cx + (-bx + ax) * cy) * inv_norm_2) *
            inv_norm * grad_normals[3 * f + 0] +
        ((-bx + ax) - cy * ((-ay + by) * cx + (-bx + ax) * cy) * inv_norm_2) *
            inv_norm * grad_normals[3 * f + 1] +
        -cz * ((-ay + by) * cx + (-bx + ax) * cy) * inv_norm_3 *
            grad_normals[3 * f + 2];
    atomicAdd(grad_verts + 3 * i0 + 2, grad_v0_z);

    // grad v1 coming from grad areas and grad normals
    const float grad_v1_x =
        (by * cz - bz * cy) / 2.0 * inv_norm * grad_areas[f] +
        -cx * (by * cz - bz * cy) * inv_norm_3 * grad_normals[3 * f + 0] +
        (-bz - cy * (by * cz - bz * cy) * inv_norm_2) * inv_norm *
            grad_normals[3 * f + 1] +
        (by - cz * (by * cz - bz * cy) * inv_norm_2) * inv_norm *
            grad_normals[3 * f + 2];
    atomicAdd(grad_verts + 3 * i1 + 0, grad_v1_x);

    const float grad_v1_y =
        (bz * cx - bx * cz) / 2.0 * inv_norm * grad_areas[f] +
        (bz - cx * (bz * cx - bx * cz) * inv_norm_2) * inv_norm *
            grad_normals[3 * f + 0] +
        -cy * (bz * cx - bx * cz) * inv_norm_3 * grad_normals[3 * f + 1] +
        (-bx - cz * (bz * cx - bx * cz) * inv_norm_2) * inv_norm *
            grad_normals[3 * f + 2];
    atomicAdd(grad_verts + 3 * i1 + 1, grad_v1_y);

    const float grad_v1_z =
        (bx * cy - by * cx) / 2.0 * inv_norm * grad_areas[f] +
        (-by - cx * (bx * cy - by * cx) * inv_norm_2) * inv_norm *
            grad_normals[3 * f + 0] +
        (bx - cx * (bx * cy - by * cx) * inv_norm_2) * inv_norm *
            grad_normals[3 * f + 1] +
        -cz * (bx * cy - by * cx) * inv_norm_3 * grad_normals[3 * f + 2];
    atomicAdd(grad_verts + 3 * i1 + 2, grad_v1_z);

    // grad v2 coming from grad areas
    const float grad_v2_x =
        (az * cy - ay * cz) / 2.0 * inv_norm * grad_areas[f] +
        -cx * (az * cy - ay * cz) * inv_norm_3 * grad_normals[3 * f + 0] +
        (az - cy * (az * cy - ay * cz) * inv_norm_2) * inv_norm *
            grad_normals[3 * f + 1] +
        (-ay - cz * (az * cy - ay * cz) * inv_norm_2) * inv_norm *
            grad_normals[3 * f + 2];
    atomicAdd(grad_verts + 3 * i2 + 0, grad_v2_x);

    const float grad_v2_y =
        (ax * cz - az * cx) / 2.0 * inv_norm * grad_areas[f] +
        (-az - cx * (ax * cz - az * cx) * inv_norm_2) * inv_norm *
            grad_normals[3 * f + 0] +
        -cy * (ax * cz - az * cx) * inv_norm_3 * grad_normals[3 * f + 1] +
        (ax - cz * (ax * cz - az * cx) * inv_norm_2) * inv_norm *
            grad_normals[3 * f + 2];
    atomicAdd(grad_verts + 3 * i2 + 1, grad_v2_y);

    const float grad_v2_z =
        (ay * cx - ax * cy) / 2.0 * inv_norm * grad_areas[f] +
        (ay - cx * (ay * cx - ax * cy) * inv_norm_2) * inv_norm *
            grad_normals[3 * f + 0] +
        (-ax - cy * (ay * cx - ax * cy) * inv_norm_2) * inv_norm *
            grad_normals[3 * f + 1] +
        -cz * (ay * cx - ax * cy) * inv_norm_3 * grad_normals[3 * f + 2];
    atomicAdd(grad_verts + 3 * i2 + 2, grad_v2_z);
  }
}

std::tuple<at::Tensor, at::Tensor> FaceAreasNormalsForwardCuda(
    const at::Tensor verts,
    const at::Tensor faces) {
  const auto V = verts.size(0);
  const auto F = faces.size(0);

  // Check inputs are on the same device
  at::TensorArg verts_t{verts, "verts", 1}, faces_t{faces, "faces", 2};
  at::CheckedFrom c = "FaceAreasNormalsForwardCuda";
  at::checkAllSameGPU(c, {verts_t, faces_t});

  // Set the device for the kernel launch based on the device of verts
  at::cuda::CUDAGuard device_guard(verts.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  at::Tensor areas = at::empty({F}, verts.options());
  at::Tensor normals = at::empty({F, 3}, verts.options());

  if (areas.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(areas, normals);
  }

  const int blocks = 64;
  const int threads = 512;

  AT_DISPATCH_FLOATING_TYPES(
      verts.scalar_type(), "face_areas_normals_forward_cuda", ([&] {
        FaceAreasNormalsForwardKernel<scalar_t><<<blocks, threads, 0, stream>>>(
            verts.contiguous().data_ptr<scalar_t>(),
            faces.contiguous().data_ptr<int64_t>(),
            areas.data_ptr<scalar_t>(),
            normals.data_ptr<scalar_t>(),
            V,
            F);
      }));
  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(areas, normals);
}

at::Tensor FaceAreasNormalsBackwardCuda(
    const at::Tensor grad_areas,
    const at::Tensor grad_normals,
    const at::Tensor verts,
    const at::Tensor faces) {
  // Check inputs are on the same device
  at::TensorArg verts_t{verts, "verts", 1}, faces_t{faces, "faces", 2},
      grad_areas_t{grad_areas, "grad_areas", 3},
      grad_normals_t{grad_normals, "grad_normals", 4};
  at::CheckedFrom c = "FaceAreasNormalsBackwardCuda";
  at::checkAllSameGPU(c, {verts_t, faces_t, grad_areas_t, grad_normals_t});

  // Set the device for the kernel launch based on the device of verts
  at::cuda::CUDAGuard device_guard(verts.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const auto V = verts.size(0);
  const auto F = faces.size(0);

  at::Tensor grad_verts = at::zeros({V, 3}, grad_areas.options());

  if (grad_verts.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return grad_verts;
  }

  const int blocks = 64;
  const int threads = 512;
  // TODO(gkioxari) add AT_DISPATCH_FLOATING_TYPES once atomicAdd supports
  // doubles. Currently, support is for floats only.
  FaceAreasNormalsBackwardKernel<<<blocks, threads, 0, stream>>>(
      grad_areas.contiguous().data_ptr<float>(),
      grad_normals.contiguous().data_ptr<float>(),
      verts.contiguous().data_ptr<float>(),
      faces.contiguous().data_ptr<int64_t>(),
      grad_verts.data_ptr<float>(),
      V,
      F);

  AT_CUDA_CHECK(hipGetLastError());
  return grad_verts;
}
